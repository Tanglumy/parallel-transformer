#include "hip/hip_runtime.h"
__global__ void gemm(const float*A, const float* B,int m,int n,int k){
    __shared__ float** result;
    x = blockIdx.x*blockDim.x+threadIdx.x;
    y = blockIdx.y*blockDim.y+threadIdx.y;
    if(x<m&&y<n){
        __shared__ float c=0
        for(int i=0;i<k;i++){
            c+= A[x*k+i]*B[i*n+y];
        }
        C[x*n+y]=c;// C[x][y]
        //优化方法：1。threadPerblock设置为32的整数倍，
}