#include "hip/hip_runtime.h"
__global__ void gemm(const float*A, const float* B,int m,int n,int k){
    x = blockIdx.x*blockDim.x+threadIdx.x;
    y = blockIdx.y*blockDim.y+threadIdx.y;
    if(x<m&&y<n){
        __shared__ float c=0.0;
        for(int i=0;i<k;i++){
            c+= A[x*k+i]*B[i*n+y];// A[x][i], B[i][y]
        }
        C[x*n+y]=c;// C[x][y]
        //优化方法：1。threadPerblock设置为32的整数倍，
}
}
__global__ void gemm_mem(const float*A, const float* B,int m,int n,int k){
    x = blockIdx.x*blockDim.x+threadIdx.x;
    y = blockIdx.y*blockDim.y+threadIdx.y;
    __shared__ nA[m][k];
    
    __shared__ nB[n][k];
    if(x<m&&y<n){
        float c=0.0;
        for(int i=0;i<k;i++){
            c+=
        }
    }
}